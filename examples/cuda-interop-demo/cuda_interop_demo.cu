#include "hip/hip_runtime.h"
#include "cuda_interop_demo.h"

#include <cuda_gl_interop.h>

#include <iostream>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

#ifdef __DRIVER_TYPES_H__
static const char *_cudaGetErrorEnum(hipError_t error) {
  return hipGetErrorName(error);
}
#endif

#ifdef CUDA_DRIVER_API
// CUDA Driver API errors
static const char *_cudaGetErrorEnum(hipError_t error) {
  static char unknown[] = "<unknown>";
  const char *ret = NULL;
  hipDrvGetErrorName(error, &ret);
  return ret ? ret : unknown;
}
#endif

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}

void printCUDAInfo()
{
    std::cout << "CUDA Info:\n";
    std::cout << "  CUDA Compiler version " << 
                 " Major: " << __CUDACC_VER_MAJOR__ <<
                 " Minor: " << __CUDACC_VER_MINOR__ <<
                 " Build: " << __CUDACC_VER_BUILD__ <<
                 std::endl;

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "  CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "  CUDA Driver version: " << driver_ver << std::endl;
}

uint32_t ceilingDivide(uint32_t a, uint32_t b) {
  return a/b + (a % b != 0);
}

std::tuple<uint32_t,uint32_t> computeBlockThreadSizes(size_t nPts) {
  uint32_t nThreads = 256;
  uint32_t nBlocks = ceilingDivide(nPts,nThreads);
  return std::tuple<uint32_t,uint32_t>(nBlocks, nThreads);
}

__global__ void initRNG_kernel(hiprandState* randState, uint32_t nPts) {
  uint32_t idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx >= nPts) return;

  hiprand_init(1234, idx, 0, &randState[idx]);
}


void initializeCUDAData(float3*& positionBuffer, hiprandState*& randState, size_t nPts) {

  // Allocate the position buffer
  std::cout << "Allocating CUDA buffer for " << nPts << " pts...\n";
  size_t nBytes = nPts*3*sizeof(float);
  checkCudaErrors(hipMalloc((void **)&positionBuffer, nBytes));
  checkCudaErrors(hipMemset(positionBuffer, 0, nBytes));
  std::cout << " ...done.\n";

  uint32_t nBlocks, nThreads;
  std::tie(nBlocks, nThreads) = computeBlockThreadSizes(nPts);
  std::cout << "nBlocks = " << nBlocks << "  nThreads = " << nThreads << std::endl;


  // Allocate & initialize the RNGs
  std::cout << "Allocating RNGs...\n";
  checkCudaErrors(hipMalloc(&randState, nPts*sizeof(hiprandState)));
  
  std::cout << "Initializing RNGs...\n";
  
  initRNG_kernel<<<nBlocks,nThreads>>>(randState, nPts);

  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipDeviceSynchronize());

  std::cout << " ...done.\n";
}

void initializeOpenGLMappedBuffer(uint32_t glTargetBuffID, hipGraphicsResource*& glResource) {
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&glResource, glTargetBuffID, hipGraphicsRegisterFlagsWriteDiscard));
}

void freeCUDAData(float3*& positionBuffer, hiprandState*& randState) {
  hipFree(positionBuffer);
  hipFree(randState);
}

void freeOpenGLMappedBuffer(hipGraphicsResource*& glResource) {
    checkCudaErrors(hipGraphicsUnregisterResource(glResource));
    glResource = nullptr;
}

__global__ void diffusePositions_kernel(float3* pos, hiprandState* randState, float stepSize, uint32_t nPts) {
    uint32_t i = threadIdx.x+blockDim.x*blockIdx.x;
    if (i >= nPts) return;

    // get the random state
    hiprandState localState = randState[i];

    float rx = hiprand_normal(&localState);
    float ry = hiprand_normal(&localState);
    float rz = hiprand_normal(&localState);

    // Update the position
    float3 p = pos[i];
    pos[i] = make_float3(p.x + rx*stepSize, p.y + ry*stepSize, p.z + rz*stepSize);
    
    // store the random state
    randState[i] = localState;
}

void diffusePositions(float3* positionBuffer, hiprandState* randState, float stepSize, size_t nPts) {
  
  //std::cout << "Diffusing positions...\n";
  
  uint32_t nBlocks, nThreads;
  std::tie(nBlocks, nThreads) = computeBlockThreadSizes(nPts);

  diffusePositions_kernel<<<nBlocks,nThreads>>>(positionBuffer, randState, stepSize, nPts);

  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipDeviceSynchronize());

  //std::cout << " ...done.\n";
}

std::vector<std::array<float,3>> getPositionsCPU(float3*& positionBuffer, size_t nPts) {
  //std::cout << "Copying position state...\n";

  hipDeviceSynchronize();

  std::vector<std::array<float,3>> positionBuffer_CPU(nPts);
  checkCudaErrors(hipMemcpy(&positionBuffer_CPU[0][0], positionBuffer, nPts*3*sizeof(float), hipMemcpyDeviceToHost));

  //std::cout << " ...done.\n";
  return positionBuffer_CPU;
}

__global__ void copyFloat3_kernel(float3* source, float3* target, uint32_t nPts) {
    uint32_t i = threadIdx.x+blockDim.x*blockIdx.x;
    if (i >= nPts) return;

    target[i] = source[i];
}

void copyPositionsToGL(float3*& positionBuffer, hipGraphicsResource*& glResource, size_t nPts) {

    // map the OpenGL buffer to a CUDA memory block
    checkCudaErrors(hipGraphicsMapResources(1, &glResource));

    float3* mappedBuff;
    size_t mappedBuffSize; // should == nPts
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&mappedBuff, &mappedBuffSize,
                                                         glResource));

    // run a simpel kernel which just copies
    uint32_t nBlocks, nThreads;
    std::tie(nBlocks, nThreads) = computeBlockThreadSizes(nPts);
    copyFloat3_kernel<<<nBlocks,nThreads>>>(positionBuffer, mappedBuff, nPts);

    // unmap the buffer now that we are done with it
    checkCudaErrors(hipGraphicsUnmapResources(1, &glResource));
}
